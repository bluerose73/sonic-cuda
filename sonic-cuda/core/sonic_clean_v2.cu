#include "hip/hip_runtime.h"
#include <sonic-cuda/core/sonic_clean_v2.h>

static float h_filter_11[11][11];
static float h_peak_filter[7][7];
__constant__ static float d_filter_11[11][11];
__constant__ static float d_peak_filter[7][7];

#define IN_TILE_WIDTH 32

#define BLUR_MASK_WIDTH 11
#define BLUR_MASK_RADIUS 5
#define BLUR_OUT_TILE_WIDTH 22

#define LOCAL_MAX_MASK_WIDTH 3
#define LOCAL_MAX_MASK_RADIUS 1
#define LOCAL_MAX_OUT_TILE_WIDTH 30

#define PEAK_MASK_WIDTH 7
#define PEAK_MASK_RADIUS 3
#define PEAK_OUT_TILE_WIDTH 26

static void fill_blur_filter_11(float filter[11][11]) {
    for (int i = 0; i < 11; ++i) {
        for (int j = 0; j < 11; ++j) {
            filter[i][j] = -1.0 / 112;
        }
    }
    
    for (int i = 4; i <= 6; ++i) {
        for (int j = 4; j <= 6; ++j) {
            filter[i][j] = 1.0 / 10;
        }
    }
    
    filter[5][5] = 1.0 / 5;
}

static void fill_peak_filter(float filter[7][7]) {
    for (int i = 0; i < 7; ++i) {
        for (int j = 0; j < 7; ++j) {
            filter[i][j] = -1.0;
        }
    }

    filter[3][3] = 1.0;
}

__global__ void blur_kernel(const float* d_data, const float* d_background,
        int height, int width, int frames, float threshold, int ignore_border_px, float* d_result) {
    __shared__ float tile_in[IN_TILE_WIDTH][IN_TILE_WIDTH];

    int f = blockIdx.z;
    int out_x = blockIdx.x * BLUR_OUT_TILE_WIDTH + threadIdx.x;
    int out_y = blockIdx.y * BLUR_OUT_TILE_WIDTH + threadIdx.y;

    int in_x = out_x - BLUR_MASK_RADIUS;
    int in_y = out_y - BLUR_MASK_RADIUS;

    if (in_x >= 0 && in_x < width && in_y >= 0 && in_y < height) {
        tile_in[threadIdx.y][threadIdx.x] = d_data[f * height * width + in_y * width + in_x];
    } else {
        tile_in[threadIdx.y][threadIdx.x] = 0;
    }
    __syncthreads();

    float sum = 0;

    if (threadIdx.x < BLUR_OUT_TILE_WIDTH && threadIdx.y < BLUR_OUT_TILE_WIDTH
            && out_x < width && out_y < height) {
        for (int i = 0; i < BLUR_MASK_WIDTH; ++i) {
            for (int j = 0; j < BLUR_MASK_WIDTH; ++j) {
                sum += tile_in[threadIdx.y + i][threadIdx.x + j] * d_filter_11[i][j];
            }
        }

        if (out_x < ignore_border_px || out_x >= width - ignore_border_px ||
            out_y < ignore_border_px || out_y >= height - ignore_border_px) {
            sum = 0;
        }

        float adjusted_threshold = threshold + 4 * sqrt(
                d_background[f * height * width + out_y * width + out_x]);
        if (sum < adjusted_threshold) {
            sum = 0;
        }

        d_result[f * height * width + out_y * width + out_x] = sum;
    }
}

__global__ void local_max_kernel(const float* d_data, int height, int width, int frames,
        float* d_result) {
    __shared__ float tile_in[IN_TILE_WIDTH][IN_TILE_WIDTH];

    int f = blockIdx.z;
    int out_x = blockIdx.x * LOCAL_MAX_OUT_TILE_WIDTH + threadIdx.x;
    int out_y = blockIdx.y * LOCAL_MAX_OUT_TILE_WIDTH + threadIdx.y;

    int in_x = out_x - LOCAL_MAX_MASK_RADIUS;
    int in_y = out_y - LOCAL_MAX_MASK_RADIUS;

    if (in_x >= 0 && in_x < width && in_y >= 0 && in_y < height) {
        tile_in[threadIdx.y][threadIdx.x] = d_data[f * height * width + in_y * width + in_x];
    } else {
        tile_in[threadIdx.y][threadIdx.x] = 0;
    }
    __syncthreads();

    if (threadIdx.x < LOCAL_MAX_OUT_TILE_WIDTH && threadIdx.y < LOCAL_MAX_OUT_TILE_WIDTH
            && out_x < width && out_y < height) {
        float center_value = tile_in[threadIdx.y + LOCAL_MAX_MASK_RADIUS][threadIdx.x + LOCAL_MAX_MASK_RADIUS];
        bool is_local_max = true;

        for (int i = 0; i < LOCAL_MAX_MASK_WIDTH; ++i) {
            for (int j = 0; j < LOCAL_MAX_MASK_WIDTH; ++j) {
                if (tile_in[threadIdx.y + i][threadIdx.x + j] > center_value) {
                    is_local_max = false;
                }
            }
        }

        if (is_local_max && in_x > 0 && in_x < width - 1 && in_y > 0 && in_y < height - 1) {
            d_result[f * height * width + out_y * width + out_x] = center_value;
        } else {
            d_result[f * height * width + out_y * width + out_x] = 0;
        }
    }
}

__global__ void find_peak_kernel(const float* d_data, int height, int width, int frames,
        int* peak_x, int* peak_y, int* peak_f, int* n_locs) {
    __shared__ float tile_in[IN_TILE_WIDTH][IN_TILE_WIDTH];

    int f = blockIdx.z;
    int out_x = blockIdx.x * PEAK_OUT_TILE_WIDTH + threadIdx.x;
    int out_y = blockIdx.y * PEAK_OUT_TILE_WIDTH + threadIdx.y;
    int in_x = out_x - PEAK_MASK_RADIUS;
    int in_y = out_y - PEAK_MASK_RADIUS;
    
    if (in_x >= 0 && in_x < width && in_y >= 0 && in_y < height) {
        tile_in[threadIdx.y][threadIdx.x] = d_data[f * height * width + in_y * width + in_x];
    } else {
        tile_in[threadIdx.y][threadIdx.x] = 0;
    }
    __syncthreads();

    if (threadIdx.x < PEAK_OUT_TILE_WIDTH && threadIdx.y < PEAK_OUT_TILE_WIDTH
            && out_x < width && out_y < height) {
        float sum = 0;
        for (int i = 0; i < PEAK_MASK_WIDTH; ++i) {
            for (int j = 0; j < PEAK_MASK_WIDTH; ++j) {
                sum += tile_in[threadIdx.y + i][threadIdx.x + j] * d_peak_filter[i][j];
            }
        }

        if (sum > 0) {
            int idx = atomicAdd(n_locs, 1);
            peak_x[idx] = out_y;
            peak_y[idx] = out_x;
            peak_f[idx] = f;
        }
    }
}

void sonic_clean_v2(const float* d_data, int height, int width, int frames,
    const float* d_background, float threshold, int ignore_border_px,
    int* d_peak_x, int* d_peak_y, int* d_peak_f, int* n_locs,
    float* blurred_data, float* local_max_data  // for debugging
) {
    fill_blur_filter_11(h_filter_11);
    fill_peak_filter(h_peak_filter);
    hipMemcpyToSymbol(HIP_SYMBOL(d_filter_11), h_filter_11, 11 * 11 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_peak_filter), h_peak_filter, 7 * 7 * sizeof(float));

    int *n_loc;
    hipMalloc(&n_loc, sizeof(int));
    hipMemset(n_loc, 0, sizeof(int));

    float *d_blurred_data, *d_local_max_data;
    hipMalloc(&d_blurred_data, frames * height * width * sizeof(float));
    hipMalloc(&d_local_max_data, frames * height * width * sizeof(float));

    dim3 block_size(IN_TILE_WIDTH, IN_TILE_WIDTH);

    dim3 blur_grid_size((width - 1) / BLUR_OUT_TILE_WIDTH + 1, (height - 1) / BLUR_OUT_TILE_WIDTH + 1, frames);
    blur_kernel<<<blur_grid_size, block_size>>>(d_data, d_background, height, width, frames, threshold, ignore_border_px, d_blurred_data);
    
    dim3 local_max_grid_size((width - 1) / LOCAL_MAX_OUT_TILE_WIDTH + 1, (height - 1) / LOCAL_MAX_OUT_TILE_WIDTH + 1, frames);
    local_max_kernel<<<local_max_grid_size, block_size>>>(d_blurred_data, height, width, frames, d_local_max_data);

    dim3 peak_grid_size((width - 1) / PEAK_OUT_TILE_WIDTH + 1, (height - 1) / PEAK_OUT_TILE_WIDTH + 1, frames);
    find_peak_kernel<<<peak_grid_size, block_size>>>(d_local_max_data, height, width, frames, d_peak_x, d_peak_y, d_peak_f, n_loc);

    if (local_max_data != nullptr) {
        hipMemcpy(local_max_data, d_local_max_data, frames * height * width * sizeof(float), hipMemcpyDeviceToHost);
    }
    if (blurred_data != nullptr) {
        hipMemcpy(blurred_data, d_blurred_data, frames * height * width * sizeof(float), hipMemcpyDeviceToHost);
    }

    hipMemcpy(n_locs, n_loc, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(n_loc);
    hipFree(d_blurred_data);
    hipFree(d_local_max_data);
}