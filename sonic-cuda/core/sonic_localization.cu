#include "hip/hip_runtime.h"
#include <sonic-cuda/core/sonic_localization.h>
#include <sonic-cuda/core/sonic_roi.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <cmath>
#include <iostream>
#include <cstdio>

// CUDA error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "sonic_localization:CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(error)); \
            goto cleanup; \
        } \
    } while(0)

// cuFFT error checking macro
#define CUFFT_CHECK(call) \
    do { \
        hipfftResult error = call; \
        if (error != HIPFFT_SUCCESS) { \
            fprintf(stderr, "sonic_localization:cuFFT error at %s:%d - %s\n", __FILE__, __LINE__, cufftGetErrorString(error)); \
            goto cleanup; \
        } \
    } while(0)

constexpr float PI = 3.14159265358979323846;

const char* cufftGetErrorString(hipfftResult error) {
    switch (error) {
        case HIPFFT_SUCCESS: return "HIPFFT_SUCCESS";
        case HIPFFT_INVALID_PLAN: return "HIPFFT_INVALID_PLAN";
        case HIPFFT_ALLOC_FAILED: return "HIPFFT_ALLOC_FAILED";
        case HIPFFT_INVALID_TYPE: return "HIPFFT_INVALID_TYPE";
        case HIPFFT_INVALID_VALUE: return "HIPFFT_INVALID_VALUE";
        case HIPFFT_INTERNAL_ERROR: return "HIPFFT_INTERNAL_ERROR";
        case HIPFFT_EXEC_FAILED: return "HIPFFT_EXEC_FAILED";
        case HIPFFT_SETUP_FAILED: return "HIPFFT_SETUP_FAILED";
        case HIPFFT_INVALID_SIZE: return "HIPFFT_INVALID_SIZE";
        case HIPFFT_UNALIGNED_DATA: return "HIPFFT_UNALIGNED_DATA";
        case HIPFFT_INCOMPLETE_PARAMETER_LIST: return "HIPFFT_INCOMPLETE_PARAMETER_LIST";
        case HIPFFT_INVALID_DEVICE: return "HIPFFT_INVALID_DEVICE";
        case HIPFFT_PARSE_ERROR: return "HIPFFT_PARSE_ERROR";
        case HIPFFT_NO_WORKSPACE: return "HIPFFT_NO_WORKSPACE";
        case HIPFFT_NOT_IMPLEMENTED: return "HIPFFT_NOT_IMPLEMENTED";
        case CUFFT_LICENSE_ERROR: return "CUFFT_LICENSE_ERROR";
        case HIPFFT_NOT_SUPPORTED: return "HIPFFT_NOT_SUPPORTED";
        default: return "Unknown CUFFT error";
    }
}

__device__ inline float fft_value_to_angle(hipfftComplex fft_value) {
    return atan2(fft_value.y, fft_value.x);
}

__global__ void fft_value_to_locations(const hipfftComplex* d_fft_values,
        const int* d_peak_x, const int* d_peak_y, int n_locs,
        float* d_loc_x, float* d_loc_y) {
    
    #define get_fft_value(n, x, y) (d_fft_values[(n) * ROI_WIDTH * (ROI_WIDTH / 2 + 1) + (x) * (ROI_WIDTH / 2 + 1) + (y)])
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n_locs) {

        hipfftComplex fft_value_x = get_fft_value(idx, 1, 0);
        float angle_x = fft_value_to_angle(fft_value_x);
        angle_x = angle_x - 2 * PI * (int) (angle_x > 0);

        // This kernel will produce the same result as the MATLAB Sonic code
        // Matlab peak uses 1-based indexing, we use 0-based indexing, so 1.0 is added to the offset
        // The Sonic localization considers the center of a 7x7 ROI as (4.0, 4.0), so additional 0.5 is added to the offset
        float offset_x = (abs(angle_x) / (2 * PI) * ROI_WIDTH) - ROI_WIDTH / 2.0 + 1.5;

        hipfftComplex fft_value_y = get_fft_value(idx, 0, 1);
        float angle_y = fft_value_to_angle(fft_value_y);
        angle_y = angle_y - 2 * PI * (int) (angle_y > 0);
        float offset_y = (abs(angle_y) / (2 * PI) * ROI_WIDTH) - ROI_WIDTH / 2.0 + 1.5;

        d_loc_x[idx] = d_peak_x[idx] + offset_x;
        d_loc_y[idx] = d_peak_y[idx] + offset_y;
    }

    #undef get_fft_value
}

int sonic_localization(const float* d_roi, const int* d_peak_x, const int* d_peak_y, int n_locs,
        float* d_loc_x, float* d_loc_y) {
    
    // Early return if no locations to process
    if (n_locs == 0) {
        return 0;
    }
    
    // Input parameter validation
    if (d_roi == nullptr) {
        fprintf(stderr, "sonic_localization:Error: d_roi is NULL\n");
        return -1;
    }
    if (d_peak_x == nullptr || d_peak_y == nullptr) {
        fprintf(stderr, "sonic_localization:Error: peak arrays are NULL\n");
        return -1;
    }
    if (d_loc_x == nullptr || d_loc_y == nullptr) {
        fprintf(stderr, "sonic_localization:Error: location arrays are NULL\n");
        return -1;
    }
    if (n_locs < 0) {
        fprintf(stderr, "sonic_localization:Error: invalid n_locs (%d)\n", n_locs);
        return -1;
    }

    // Initialize all variables at the beginning to avoid goto bypass issues
    hipfftHandle plan = 0;
    hipfftComplex* fft_values = nullptr;
    int dims[2] = {ROI_WIDTH, ROI_WIDTH};
    int block_size = 256;
    int grid_size;
    bool plan_created = false;
    
    CUDA_CHECK(hipMalloc((void**)&fft_values, n_locs * ROI_WIDTH * (ROI_WIDTH / 2 + 1) * sizeof(hipfftComplex)));

    CUFFT_CHECK(hipfftPlanMany(&plan, 2, dims, NULL, 0, 0, NULL, 0, 0, HIPFFT_R2C, n_locs));
    plan_created = true;
    
    CUFFT_CHECK(hipfftExecR2C(plan, (hipfftReal*)d_roi, fft_values));

    grid_size = (n_locs + block_size - 1) / block_size;
    fft_value_to_locations<<<grid_size, block_size>>>(fft_values, d_peak_x, d_peak_y, n_locs, d_loc_x, d_loc_y);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Success - clean up and return
    if (fft_values) hipFree(fft_values);
    if (plan_created) hipfftDestroy(plan);
    
    return 0;

cleanup:
    // Error occurred - free all allocated resources
    if (fft_values) hipFree(fft_values);
    if (plan_created) hipfftDestroy(plan);
    
    return -1;
}